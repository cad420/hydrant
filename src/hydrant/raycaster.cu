#include "hip/hip_runtime.h"
#include "raycaster.hpp"

VM_BEGIN_MODULE( hydrant )

__global__ void
  cast_kernel_impl( CastOptions opts )
{
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if ( x >= opts.resolution.x || y >= opts.resolution.y ) {
		return;
	}

  auto cc = vec2( opts.resolution ) / 2.f;
  auto uv = ( vec2{ x, y } - cc ) * 2.f / float( opts.resolution.y );
  Ray ray = { 
    opts.ray_o, 
    normalize( vec3( opts.trans * vec4( uv.x, -uv.y, -opts.itg_fovy, 1 ) ) - opts.ray_o )
  };

  opts.shader( ray, opts.image + opts.pixel_size * ( opts.resolution.x * y + x ) );
}

CUFX_DEFINE_KERNEL( cast_kernel, cast_kernel_impl );

VM_END_MODULE()
