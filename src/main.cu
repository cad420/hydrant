#include "hip/hip_runtime.h"
#include <fstream>
#include <atomic>
#include <cstdlib>
#include <cppfs/fs.h>
#include <cppfs/FileHandle.h>
#include <cppfs/FilePath.h>
#include <VMUtils/timer.hpp>
#include <VMUtils/cmdline.hpp>
#include <VMUtils/fmt.hpp>
#include <cudafx/device.hpp>
#include <cudafx/transfer.hpp>
#include <varch/utils/io.hpp>
#include <varch/unarchive/unarchiver.hpp>
#include <varch/thumbnail.hpp>
#include <varch/package_meta.hpp>
#include <hydrant/buffer3d.hpp>
#include <hydrant/raycaster.hpp>
#include "shaders/volume_shader.hpp"

using namespace std;
using namespace vol;
using namespace hydrant;
using namespace cppfs;

inline void ensure_dir( std::string const &path_v )
{
	auto path = cppfs::fs::open( path_v );
	if ( !path.exists() ) {
		vm::eprintln( "the specified path '{}' doesn't exist",
					  path_v );
		exit( 1 );
	} else if ( path.isFile() ) {
		vm::eprintln( "the specified path '{}' is not a file",
					  path_v );
		exit( 1 );
	}
}

int main( int argc, char **argv )
{
	cmdline::parser a;
	a.add<string>( "in", 'i', "input directory", true );
	a.add<string>( "out", 'o', "output filename", true );
	a.add( "thumb", 't', "take snapshots of single thumbnail file" );
	a.add<string>( "config", 'c', "config file", false );
	a.add<float>( "x", 'x', "camera.x", false, 3 );
	a.add<float>( "y", 'y', "camera.y", false, 2 );
	a.add<float>( "z", 'z', "camera.z", false, 2 );

	a.parse_check( argc, argv );

	auto in = FilePath( a.get<string>( "in" ) );
	ensure_dir( in.resolved() );
	auto out = FilePath( a.get<string>( "out" ) );
	auto device = cufx::Device::scan()[ 0 ];

	PackageMeta meta;
	ifstream meta_is( in.resolve( "package_meta.json" ).resolved() );
	meta_is >> meta;

	using Shader = VolumeRayEmitShader;
	Shader shader;

	cufx::Image<typename Shader::Pixel> image( 512, 512 );
	auto device_swap = device.alloc_image_swap( image );
	auto img_view = image.view().with_device_memory( device_swap.second );
	img_view.copy_to_device().launch();

	/* input file */

	auto &lvl0_arch = meta.sample_levels[ 0 ].archives[ 0 ];
	ifstream is( in.resolve( lvl0_arch.path ).resolved(), ios::ate | ios::binary );
	auto len = is.tellg();
	StreamReader reader( is, 0, len );
	Unarchiver unarchiver( reader );
	Thumbnail chebyshev( in.resolve( lvl0_arch.thumbnails[ "chebyshev" ] ).resolved() );

	glm::uvec3 dim = { unarchiver.dim().x, unarchiver.dim().y, unarchiver.dim().z };
	glm::uvec3 bdim = { unarchiver.padded_block_size(), unarchiver.padded_block_size(), unarchiver.padded_block_size() };

	/* view */
#pragma region

	glm::vec3 raw = { unarchiver.raw().x, unarchiver.raw().y, unarchiver.raw().z };
	glm::vec3 f_dim = raw / float( unarchiver.block_size() );
	// glm::vec3 max = { dim.x, dim.y, dim.z };
	auto exhibit = Exhibit{}
					 .set_center( f_dim / 2.f )
					 .set_size( f_dim );

	shader.bbox = Box3D{ { 0, 0, 0 }, f_dim };
	shader.step = 1e-2f * f_dim.x / 4.f;
	shader.cache_du.x = float( unarchiver.padding() ) / unarchiver.block_size();
	shader.cache_du.y = float( unarchiver.block_size() ) / unarchiver.padded_block_size();

	auto camera = Camera{};
	if ( a.exist( "config" ) ) {
		auto cfg = a.get<string>( "config" );
		camera = Camera::from_config( cfg );
	} else {
		auto x = a.get<float>( "x" );
		auto y = a.get<float>( "y" );
		auto z = a.get<float>( "z" );
		camera.set_position( x, y, z );
	}

#pragma endregion

	/* chebyshev texture */
#pragma region

	auto thumbnail_extent = cufx::Extent{}
							  .set_width( dim.x )
							  .set_height( dim.y )
							  .set_depth( dim.z );
	auto thumbnail_view_info = cufx::MemoryView2DInfo{}
								 .set_stride( dim.x * sizeof( float ) )
								 .set_width( dim.x )
								 .set_height( dim.y );
	auto chebyshev_arr = device.alloc_arraynd<float, 3>( thumbnail_extent );
	// vm::println( "dim = {}, thumbnail_extent = {}", dim, thumbnail_extent );
	cufx::MemoryView3D<float> chebyshev_view( chebyshev.data(), thumbnail_view_info, thumbnail_extent );
	cufx::memory_transfer( chebyshev_arr, chebyshev_view ).launch();
	cufx::Texture chebyshev_texture( chebyshev_arr,
									 cufx::Texture::Options::as_array()
									   .set_address_mode( cufx::Texture::AddressMode::Clamp ) );
	shader.chebyshev_tex = chebyshev_texture;

#pragma endregion

	/* present texture */
#pragma region

	Buffer3D<int> present_buf( dim );
	auto present_extent = cufx::Extent{}
							.set_width( dim.x )
							.set_height( dim.y )
							.set_depth( dim.z );
	auto present_arr = device.alloc_arraynd<int, 3>( present_extent );
	// vm::println( "dim = {}, present_extent = {}", dim, present_extent );
	auto present_view_info = cufx::MemoryView2DInfo{}
							   .set_stride( dim.x * sizeof( int ) )
							   .set_width( dim.x )
							   .set_height( dim.y );
	cufx::MemoryView3D<int> present_view( present_buf.data(), present_view_info, present_extent );
	// cufx::Texture present_texture( present_arr, cufx::Texture::Options::as_array() );
	// cufx::memory_transfer( present_arr, present_view ).launch();
	// cufx::Texture present_texture( present_arr, cufx::Texture::Options::as_array() );
	// shader.present_tex = present_texture;

#pragma endregion

	/* absent buffer */
#pragma region

	auto wg_cnt = 32 * 32;
	shader.wg_max_emit_cnt = 8;
	shader.wg_len_bytes = sizeof( int ) +
						  shader.wg_max_emit_cnt * sizeof( glm::uvec3 );
	auto absent_glob = device.alloc_global( shader.wg_len_bytes * wg_cnt );
	vector<char> absent( absent_glob.size() );
	shader.absent_buf = absent_glob.view_1d<char>( absent_glob.size() );

#pragma endregion

	/* block buffer */

	auto pad_bs = unarchiver.padded_block_size();
	auto block_bytes = pad_bs * pad_bs * pad_bs;
	auto block_glob = device.alloc_global( block_bytes );
	auto block_view_1d = block_glob.view_1d<unsigned char>( block_bytes );
	auto block_view_info = cufx::MemoryView2DInfo{}
							 .set_stride( pad_bs * sizeof( unsigned char ) )
							 .set_width( pad_bs )
							 .set_height( pad_bs );
	auto block_extent = cufx::Extent{}
						  .set_width( pad_bs )
						  .set_height( pad_bs )
						  .set_depth( pad_bs );
	auto block_view_3d = block_glob.view_3d<unsigned char>( block_view_info, block_extent );
	vector<cufx::Array3D<unsigned char>> cache_block_arr;
	for ( int i = 0; i != MAX_CACHE_SIZE; ++i ) {
		cache_block_arr.emplace_back( device.alloc_arraynd<unsigned char, 3>( block_extent ) );
	}
	// cufx::MemoryView3D<int> chebyshev_view( chebyshev.data(), thumbnail_view_info, thumbnail_extent );
	// cufx::memory_transfer( sampler_arr, block_view_3d, hipPos{ 0, 0, 0 } ).launch();
	// cufx::Texture sampler_texture( sampler_arr, cufx::Texture::Options::as_array() );

	std::vector<vol::Idx> block_idxs;
	chebyshev.iterate_3d(
	  [&]( vol::Idx const &idx ) {
		  if ( !chebyshev[ idx ] ) {
			  block_idxs.emplace_back( idx );
		  }
	  } );
	vm::println( "{}", block_idxs );
	vector<glm::vec3> block_ccs( block_idxs.size() );
	std::transform( block_idxs.begin(), block_idxs.end(), block_ccs.begin(),
					[]( Idx const &idx ) { return glm::vec3( idx.x, idx.y, idx.z ) + 0.5f; } );
	vector<int> pidx( block_idxs.size() );
	for ( int i = 0; i != pidx.size(); ++i ) { pidx[ i ] = i; }
	vm::println( "{}", block_idxs.size() );

	auto et = exhibit.get_matrix();

	Raycaster raycaster;
	// raycaster.cast( exhibit, camera, img_view, shader );
	int nframes = 1;
	while ( nframes-- ) {
		std::size_t ns = 0, ns1 = 0;

		vm::Timer::Scoped timer( [&]( auto dt ) {
			vm::println( "time: {} / {} / {}", dt.ms(),
						 ns / 1000 / 1000,
						 ns1 / 1000 / 1000 );
		} );

		glm::vec3 cp = et * glm::vec4( camera.position, 1 );

		std::sort( pidx.begin(), pidx.end(),
				   [&]( int x, int y ) {
					   return glm::distance( block_ccs[ x ], cp ) <
							  glm::distance( block_ccs[ y ], cp );
				   } );

		for ( int i = 0; i < pidx.size(); i += MAX_CACHE_SIZE ) {
			vector<Idx> idxs;
			for ( int j = i; j < i + MAX_CACHE_SIZE && j < pidx.size(); ++j ) {
				idxs.emplace_back( block_idxs[ pidx[ j ] ] );
			}

			int nbytes = 0, blkid = 0;
			vector<cufx::Texture> cache_texs;
			memset( present_buf.data(), -1, present_buf.bytes() );

			{
				vm::Timer::Scoped timer( [&]( auto dt ) {
					ns += dt.ns().cnt();
				} );

				unarchiver.unarchive(
				  idxs,
				  [&]( Idx const &idx, VoxelStreamPacket const &pkt ) {
					  pkt.append_to( block_view_1d );
					  nbytes += pkt.length;
					  if ( nbytes >= block_bytes ) {
						  cufx::memory_transfer( cache_block_arr[ blkid ], block_view_3d ).launch();
						  //   if ( blkid == 0 ) {
						  cache_texs.emplace_back( cache_block_arr[ blkid ],
												   cufx::Texture::Options{}
													 .set_address_mode( cufx::Texture::AddressMode::Wrap )
													 .set_filter_mode( cufx::Texture::FilterMode::Linear )
													 .set_read_mode( cufx::Texture::ReadMode::NormalizedFloat )
													 .set_normalize_coords( true ) );
						  present_buf[ glm::vec3( idx.x, idx.y, idx.z ) ] = blkid;
						  nbytes = 0;
						  blkid += 1;
						  //   }
					  }
				  } );
			}

			cufx::memory_transfer( present_arr, present_view ).launch();
			cufx::Texture present_texture( present_arr, cufx::Texture::Options::as_array() );
			shader.present_tex = present_texture;

			// vm::println( "{}", cache_texs.size() );
			for ( int j = 0; j != cache_texs.size(); ++j ) {
				shader.cache_tex[ j ] = cache_texs[ j ];
			}

			{
				vm::Timer::Scoped timer( [&]( auto dt ) {
					ns1 += dt.ns().cnt();
				} );

				if ( i == 0 ) {
					raycaster.cast( exhibit, camera, img_view, shader );
				} else {
					raycaster.cast( img_view, reinterpret_cast<VolumePixelShader &>( shader ) );
				}
			}
		}
	}

	img_view.copy_from_device().launch();

	image.dump( out.resolved() );
}
